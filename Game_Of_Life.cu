#include "hip/hip_runtime.h"
// This file calls our main function. It contains all OpenGL source code. It has an object of
// Game_Of_Life which it uses to determine the color of squares.

#include <stdbool.h>
#include <stdio.h>
#include <GL/glew.h>
#include <GL/freeglut.h>

#define WIDTH 1024
#define HEIGHT 768
#define NUM_SPECIES 10

enum Species { S0, S1, S2, S3, S4, S5, S6, S7, S8, S9, DEAD };
static const enum Species species_map[] = { S0, S1, S2, S3, S4, S5, S6, S7, S8, S9, DEAD };

enum Species *grid;

void set_cell(int x, int y, enum Species s) {
    int index = WIDTH * y + x;
    grid[index] = s;
}

enum Species get_cell(int x, int y) {
    int index = WIDTH * y + x;
    return grid[index];
}

void initialize_grid() {
    srand(time(NULL));
    grid = (enum Species*) malloc(WIDTH * HEIGHT * sizeof(enum Species));

    for (int i = 0; i < HEIGHT; i++) {
        for (int j = 0; j < WIDTH; j++) {
            set_cell(j, i, DEAD);
        }
    }

    for (int i = 0; i < NUM_SPECIES; i++) {
        enum Species species = species_map[i];

        int square_size = WIDTH * .10;

        // fill ~20% of square
        int number_of_squares = (int) floor((square_size * square_size) * 0.20);
        int distance_from_edge = square_size + 2;

        // choose random target on board, at least specified distance from edges
        int x_target = (rand() % (WIDTH - (distance_from_edge * 2 - 1))) + distance_from_edge;
        int y_target = (rand() % (HEIGHT - (distance_from_edge * 2 - 1))) + distance_from_edge;

        set_cell(x_target, y_target, species);

        // pick number_of_squares within (square_size x square_size) square centered on target
        int rand_x;
        int rand_y;
        for (int i = 0; i < number_of_squares; i++) {
            rand_x = x_target + ((rand() % (square_size + 1)) - (square_size/2));
            rand_y = y_target + ((rand() % (square_size + 1)) - (square_size/2));
            set_cell(rand_x, rand_y, species);
        }
    }
}

__device__ void set_d_cell(int idx, enum Species s, enum Species *grid_d) {
    grid_d[idx] = s;
}

__device__ enum Species get_d_cell(int idx, enum Species *grid_d) {
    return grid_d[idx];
}

__device__ enum Species get_d_cell(int x, int y, enum Species *grid_d) {
    int idx = y * 1024 + x;
    return grid_d[idx];
}

__device__ int number_of_neighbors(int idx, enum Species s, enum Species *grid_d) {
    int count = 0;

    int x = idx % 1024;
    int y = idx / 1024;

    // iterate over 3x3 grid centered on (x,y)
    for (int i = x - 1; i <= x + 1; i++) {
        for (int j = y - 1; j <= y + 1; j++) {
            // check only if cell isn't current cell (x,y) AND cell is not out of bounds
            if ((i != x || j != y) && (i >= 0 && j >= 0 && i < 1024 && j < 768)) {
                if (get_d_cell(i, j, grid_d) == s) {
                    count++;
                }
            }
        }
    }

    return count;
}

__device__ bool has_three_neighbors(int idx, enum Species *grid_d) {
    int count = 0;

    int x = idx % 1024;
    int y = idx / 1024;

    for (int i = x - 1; i <= x + 1; i++) {
        for (int j = y - 1; j <= y + 1; j++) {
            // check only if cell isn't current cell (x,y) AND cell is not out of bounds
            if ((i != x || j != y) && (i >= 0 && j >= 0 && i < WIDTH && j < HEIGHT)) {
                if (get_d_cell(i, j, grid_d) != DEAD) {
                    if (++count == 3) {
                        return true;
                    }
                }
            }
        }
    }
    return false;
}

__global__ void kernel(enum Species *grid_d, enum Species *species_map_d, enum Species *update_list_d) {

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < WIDTH * HEIGHT) {

        // determine species of cell
        enum Species species = get_d_cell(idx, grid_d);

        // if species lives in cell, count # neighbors and add to kill list if applicable
        if (species != DEAD) {
            int num_neighbors = number_of_neighbors(idx, species, grid_d);
            if (num_neighbors < 2 || num_neighbors > 3) {
                set_d_cell(idx, DEAD, update_list_d);
            }
        // if no species in cell, check if any species should be spawned there
        // do "heavier" get_spawn_type() only if 3 neighbors exist
        } else if (has_three_neighbors(idx, grid_d)) {
            for (int i = 0; i < 10; i++) {
                enum Species s = species_map_d[i];
                int num_neighbors = number_of_neighbors(idx, s, grid_d);
                if (s != DEAD && num_neighbors == 3) {
                    set_d_cell(idx, s, update_list_d);
                }
            }
        }
    }
}

void update_grid() {

    enum Species *grid_d;
    enum Species *update_list_d;
    enum Species *species_map_d;

    int grid_size = WIDTH * HEIGHT * sizeof(enum Species);
    int map_size = sizeof(species_map);
    hipMalloc((void **) &grid_d, grid_size);
    hipMalloc((void **) &update_list_d, grid_size);
    hipMalloc((void **) &species_map_d, map_size);
    hipMemcpy(grid_d, grid, grid_size, hipMemcpyHostToDevice);
    hipMemcpy(update_list_d, grid, grid_size, hipMemcpyHostToDevice);
    hipMemcpy(species_map_d, species_map, map_size, hipMemcpyHostToDevice);

    int block_size = 512;
    dim3 dimBlock(block_size);
    dim3 dimGrid(ceil(WIDTH * HEIGHT / (float) block_size));

    kernel<<<dimGrid, dimBlock>>>(grid_d, species_map_d, update_list_d);
    grid_d = update_list_d;

    hipMemcpy(grid, grid_d, grid_size, hipMemcpyDeviceToHost);
    hipFree(grid_d);
}

// sets the color that OpenGL will draw with
void set_color(enum Species species) {
    switch(species) {
        case S0:	glColor3f(1.0f, 0.0f, 0.0f); break; // RED
        case S1:	glColor3f(0.0f, 1.0f, 0.0f); break; // GREEN
        case S2:	glColor3f(0.1f, 0.2f, 1.0f); break; // BLUE
        case S3:	glColor3f(1.0f, 1.0f, 0.0f); break; // YELLOW
        case S4:	glColor3f(1.0f, 0.0f, 1.0f); break; // MAGENTA
        case S5:	glColor3f(0.0f, 1.0f, 1.0f); break; // CYAN
        case S6:	glColor3f(1.0f, 1.0f, 1.0f); break; // WHITE
        case S7:	glColor3f(1.0f, 0.5f, 0.0f); break; // ORANGE
        case S8:	glColor3f(0.5f, 0.5f, 0.5f); break; // GREY
        case S9:	glColor3f(0.4f, 0.0f, 1.0f); break; // VIOLET
        default:	glColor3f(0.0f, 0.0f, 0.0f);		// BLACK
    }
}

// places a square at (x, y). Must be nested in glBegin() <-> glEnd() tags
void draw_square(int x, int y) {
    if (x < 0 || y < 0 || x >= WIDTH || y >= HEIGHT) {
        printf("Invalid range in draw_square. (%d, %d) out of range", x, y);
        exit(1);
    }

    glVertex2f(x, y);
    glVertex2f(x + 1, y);
    glVertex2f(x + 1, y + 1);
    glVertex2f(x, y + 1);
}

void draw_board() {
    glBegin(GL_QUADS);
    for (int i = 0; i < HEIGHT; i++) {
        for (int j = 0; j < WIDTH; j++) {
            enum Species s = get_cell(j, i);
            set_color(s);
            draw_square(j, i);
        }
    }
    glEnd();
    glFlush();
}

// infinite loop. It fetches and operates on the cells that need to be changed forever
void display() {
    initialize_grid();
    draw_board();

    int count = 0;
    clock_t start = clock();

    for(;;) {
        update_grid();
        draw_board();

        count++;
        double duration = double(clock() - start) / CLOCKS_PER_SEC;
        if (duration > 2) {
            printf("FPS = %.2f\n", count/duration);
            start = clock();
            count = 0;
        }
    }
}

// intializate OpenGL and begin our display loop
int main(int argc, char **argv) {
    glutInit(&argc, argv);
    glutInitDisplayMode(GLUT_SINGLE | GLUT_RGB | GLUT_DEPTH);

    glutInitWindowSize(WIDTH, HEIGHT);
    glutCreateWindow("Game of Life 2.1");

    glMatrixMode(GL_PROJECTION);
    glLoadIdentity();
    glOrtho(0.0f, WIDTH, HEIGHT, 0.0f, -1.0f, 1.0f);

    glutDisplayFunc(display);

    glutMainLoop();

    free(grid);
}
